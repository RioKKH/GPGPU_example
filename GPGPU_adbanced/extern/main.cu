#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "lib.h"

// extern __constant__ float a;

int main(void)
{
	float host_a = 5.0f;
	set(host_a);
	// hipMemcpyToSymbol(HIP_SYMBOL(a), &host_a, sizeof(float));
	hipDeviceSynchronize();

	show<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
