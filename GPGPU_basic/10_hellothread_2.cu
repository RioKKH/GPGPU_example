
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void hello()
{
    printf("Hello Thread\n");
}

int main(void)
{
    hello<<<8, 1>>>();
    hipDeviceSynchronize();

    std::cout << std::endl;

    hello<<<1, 8>>>();
    hipDeviceSynchronize();

    std::cout << std::endl;

    hello<<<4, 2>>>();
    hipDeviceSynchronize();
    return 0;
}

