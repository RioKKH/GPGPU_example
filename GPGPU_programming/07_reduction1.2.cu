
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N (512)
#define Nbytes (N*sizeof(int))
#define NT (N)
#define NB (N / NT)
#define STEP (9) // reductionの段数


__global__ void reduction1(int *idata, int *odata)
{
    // スレッドと配列の要素の対応
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // スレッド番号
    int tx = threadIdx.x;
    // stepはループのカウンターとしてのみ利用しているが、
    // ループカウンターはstrideで代用可能
    // stepはコメントアウトしてしまう
    // int step; // reductionの段数をカウントする変数
    int stride; // 隣の配列要素まで距離
    
    stride = 1;
    // ストライドを2倍し、ストライドがN/2以下ならループを継続
    // <<= : シフト演算の代入演算子 a <<= 1 --> a = a << 1と同じ
    // 最終stepではstrideが配列要素数のN/2となるので、strideがN/2
    // より大きくなるとループを中断
    for (stride = 1; stride <= blockDim.x/2; stride <<= 1)
    {
        // 処理を行うスレッドを選択
        if (tx % (2 * stride) == 0)
        {
            idata[i] = idata[i] + idata[i + stride];
        }
        __syncthreads(); // スレッド間の同期を取る
        // stride = stride * 2; // ストライドを2倍して次のstepに備える
    }
    if (tx == 0) // スレッド0が総和を出力用変数odataに書き込んで終了
    {
        odata[0] = idata[0];
    }
}

void init(int *idata)
{
    int i;
    for (i=0; i<N; ++i)
    {
        idata[i] = 1;
    }
}

int main()
{
    // GPU用変数 idata: 入力、odata: 出力(総和)
    int *idata, *odata;

    // CPU用変数 host_idata: 初期化用、sum: 総和
    int *host_idata, sum;

    hipMalloc((void **)&idata, Nbytes);
    hipMalloc((void **)&odata, sizeof(int));

    // CPU側でデータを初期化してGPUへコピー
    host_idata = (int *)malloc(Nbytes);
    init(host_idata);
    hipMemcpy(idata, host_idata, Nbytes, hipMemcpyHostToDevice);
    free(host_idata);

    reduction1<<<NB, NT>>>(idata, odata);

    // GPUから総和の結果を受け取って画面表示
    hipMemcpy(&sum, odata, sizeof(int), hipMemcpyDeviceToHost);

    printf("sum = %d\n", sum);
    hipFree(idata);
    hipFree(odata);
    return 0;
}
