
#include <hip/hip_runtime.h>
#define SIZE 4096
#define THREADX 256
#define THREADY 1
#define BLOCKX (SIZE / THREADX)
#define BLOCKY (SIZE / THREADY)

__global__ void matmulGPU(float *A, float *B, float *C)
{
	int i, j, k;
	// レジスタを使う
	float sum = 0.0f;
	int tx;

	// 共有メモリを使う
	__shared__ float sB[THREADX];

	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	tx = threadIdx.x;

	for (k = 0; k < SIZE; k += THREADX)
	{
		sB[tx] = B[(k + tx) + SIZE * j];
		__syncthreads();
		for (int w = 0; w < THREADX; w++)
		{
			sum += A[i + SIZE * (k + w)] * sB[w];
		}
		__syncthreads();
	}
	C[i + SIZE * j] = sum;
}

