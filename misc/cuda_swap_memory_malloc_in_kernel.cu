#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

void showPopulation(int *population, int N)
{
    for (int i = 0; i < N; ++i)
    {
        printf("%d", population[i]);
    }
    printf("\n");
}

__global__
void swapPopulation(int *parent, int *offspring)
{
    int *temp;
    temp = parent;
    parent = offspring;
    offspring = temp;
}


int main()
{
    int *parent_host, *offspring_host;
    int *parent_dev, *offspring_dev, *temp_dev;

    float elapsed_time = 0.0f;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int N = 256000;
    int nBytes = N * sizeof(float);
    
    parent_host = (int *)malloc(nBytes);
    offspring_host = (int *)malloc(nBytes);

    hipMalloc((void **)&parent_dev, nBytes);
    hipMalloc((void **)&offspring_dev, nBytes);
    // cudaMalloc((void **)&temp_dev, nBytes);

    // initialize host data
    for (int i = 0; i < N; ++i)
    {
        parent_host[i] = i;
        offspring_host[i] = i * 10;
    }

    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    hipEventRecord(start, 0);
    // Host to Device
    hipMemcpy(parent_dev,    parent_host,    nBytes, hipMemcpyHostToDevice);
    hipMemcpy(offspring_dev, offspring_host, nBytes, hipMemcpyHostToDevice);

    // swap memories between parents and offsprings
    // temp_dev = parent_dev;
    // parent_dev = offspring_dev;
    // offspring_dev = temp_dev;
    swapPopulation<<<1, 1>>>(parent_dev, offspring_dev);
    hipDeviceSynchronize();

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);

    // Device to Host
    hipMemcpy(parent_host,    parent_dev,    nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(offspring_host, offspring_dev, nBytes, hipMemcpyDeviceToHost);

    printf("ElapsedTime: %f\n", elapsed_time);
    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    // free memories
    free(parent_host);
    free(offspring_host);
    hipFree(parent_dev);
    hipFree(offspring_dev);
}

