#include "hip/hip_runtime.h"
// cuda
#include <hip/hip_runtime.h>
#include <>

// thrust
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>

// c++
#include <iostream>

template <class T>
__global__ void plus_kernel(const T* g_x, const T* g_y, T* g_o)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    g_o[i] = g_x[i] + g_y[i];
}

int main(void)
{
    const int blocks = 1;
    const int threads = 32;
    const int N = blocks * threads;

    thrust::counting_iterator<int> ci(1);
    thrust::device_vector<int> d_x(ci, ci+N);
    thrust::device_vector<int> d_y(ci, ci+N);
    thrust::device_vector<int> d_o(blocks * threads);

    plus_kernel<int><<<blocks, threads>>>(
            thrust::raw_pointer_cast(d_x.data()),
            thrust::raw_pointer_cast(d_y.data()),
            thrust::raw_pointer_cast(d_o.data())
            );

    thrust::copy(thrust::host, d_o.cbegin(), d_o.cend(), std::ostream_iterator<int>(std::cout, " "));

    return 0;
}
