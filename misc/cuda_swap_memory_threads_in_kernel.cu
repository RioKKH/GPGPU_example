#include <iostream>
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

void showPopulation(int *population, int N)
{
    for (int i = 0; i < N; ++i)
    {
        printf("%d", population[i]);
    }
    printf("\n");
}

__global__
void swapPopulation(int *parent, int *offspring)
{
    std::uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx == 1)
    {
        for (int i = 0; i < 256000; ++i)
        {
            parent[i] = offspring[i];
        }
    }
}


int main()
{
    int *parent_host, *offspring_host;
    int *parent_dev, *offspring_dev, *temp_dev;

    float elapsed_time = 0.0f;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int N = 256000;
    int nBytes = N * sizeof(float);
    
    parent_host = (int *)malloc(nBytes);
    offspring_host = (int *)malloc(nBytes);

    hipMalloc((void **)&parent_dev, nBytes);
    hipMalloc((void **)&offspring_dev, nBytes);
    // cudaMalloc((void **)&temp_dev, nBytes);

    // initialize host data
    for (int i = 0; i < N; ++i)
    {
        parent_host[i] = i;
        offspring_host[i] = i * 10;
    }

    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    hipEventRecord(start, 0);
    // Host to Device
    hipMemcpy(parent_dev,    parent_host,    nBytes, hipMemcpyHostToDevice);
    hipMemcpy(offspring_dev, offspring_host, nBytes, hipMemcpyHostToDevice);

    dim3 blocks;
    dim3 threads;
    const int WARP = 32;
    blocks.x = 1;
    // blocks.x = N / 32;
    blocks.y = 1;
    blocks.z = 1;
    threads.x = 1;
    threads.y = 1;
    threads.z = 1;
    // swap memories between parents and offsprings
    // temp_dev = parent_dev;
    // parent_dev = offspring_dev;
    // offspring_dev = temp_dev;
    swapPopulation<<<blocks, threads>>>(parent_dev, offspring_dev);
    hipDeviceSynchronize();

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);

    // Device to Host
    hipMemcpy(parent_host,    parent_dev,    nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(offspring_host, offspring_dev, nBytes, hipMemcpyDeviceToHost);

    printf("ElapsedTime: %f\n", elapsed_time);
    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    // free memories
    free(parent_host);
    free(offspring_host);
    hipFree(parent_dev);
    hipFree(offspring_dev);
}

