#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#define CUDA_CHECK_RETURN(value)                             \
{                                                            \
    hipError_t _m_cudaStat = value;                         \
    if (_m_cudaStat != hipSuccess)                          \
    {                                                        \
        fprintf(stderr, "Error %s at line %d in file %s\n",  \
                hipGetErrorString(_m_cudaStat),             \
                __LINE__, __FILE__);                         \
        exit(1);                                             \
    }                                                        \
}

void showPopulation(int *population, int N)
{
    for (int i = 0; i < N; ++i)
    {
        printf("%d", population[i]);
    }
    printf("\n");
}


int main()
{
    int *parent_host, *offspring_host;
    int *parent_dev, *offspring_dev, *temp_dev;

    float elapsed_time = 0.0f;
    hipEvent_t start, end;
    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&end));

    int N = 256000;
    int nBytes = N * sizeof(float);
    
    parent_host = (int *)malloc(nBytes);
    offspring_host = (int *)malloc(nBytes);

    CUDA_CHECK_RETURN(hipMalloc((void **)&parent_dev, nBytes));
    CUDA_CHECK_RETURN(hipMalloc((void **)&offspring_dev, nBytes));
    // cudaMalloc((void **)&temp_dev, nBytes);

    // initialize host data
    for (int i = 0; i < N; ++i)
    {
        parent_host[i] = i;
        offspring_host[i] = i * 10;
    }

    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    hipEventRecord(start, 0);
    // Host to Device
    hipMemcpy(parent_dev,    parent_host,    nBytes, hipMemcpyHostToDevice);
    hipMemcpy(offspring_dev, offspring_host, nBytes, hipMemcpyHostToDevice);

    // swap memories between parents and offsprings
    temp_dev = parent_dev;
    parent_dev = offspring_dev;
    offspring_dev = temp_dev;
    hipDeviceSynchronize();

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);

    // Device to Host
    hipMemcpy(parent_host,    parent_dev,    nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(offspring_host, offspring_dev, nBytes, hipMemcpyDeviceToHost);

    printf("ElapsedTime: %f\n", elapsed_time);
    printf("\npost-parent\n");
    // showPopulation(parent_host, N);
    printf("\npost-offspring\n");
    // showPopulation(offspring_host, N);

    // free memories
    free(parent_host);
    free(offspring_host);
    hipFree(parent_dev);
    hipFree(offspring_dev);
}

