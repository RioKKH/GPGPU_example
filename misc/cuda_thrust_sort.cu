#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <cstdio>
#include <numeric>

// cuda
#include <hip/hip_runtime.h>
#include <>

// thrust
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>


template <typename T, int N>
__global__ void reduction1_kernel(const T* g_x, T* g_o)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    // ブロック内のスレッド数分の共有メモリ領域を確保
    // 共有メモリはブロック内のスレッドが共有出来るメモリ
    __shared__ T s_x[N];
    // グローバルメモリg_xから共有メモリs_xへ値を転送する
    s_x[tid] = (i < N) ? g_x[i] : T{};
    __syncthreads(); // 同一ブロク内のスレッドに対するバリア

    // 共有メモリの0番目に合計された値が格納されている
    for (int s = 1; s < blockDim.x; s *= 2)
    {
        if ((tid % (2 * s)) == 0)
        {
            s_x[tid] += s_x[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_o[blockIdx.x] = s_x[0];
    }
}


__global__ void pseudo_elisism(const int* dev)
{
    int numOfEliteIdx    = blockIdx.x;
    int localFitnessIdx  = threadIdx.x;
    int globalFitnessIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const int OFFSET     = blockDim.x;

    extern __shared__ volatile int s_fitness[];

    s_fitness[localFitnessIdx] = dev[globalFitnessIdx];
    s_fitness[localFitnessIdx + OFFSET] = globalFitnessIdx;
    __syncthreads();


}

// show fit and id data on CPU
void show_host(thrust::host_vector<int> host_id,
               thrust::host_vector<int> host_fit)
{
    if (host_id.size() == host_fit.size())
    {
        for (int i = 0; i < host_id.size(); ++i)
        {
            printf("%d,%d\n", host_id[i], host_fit[i]);
        }
    }
}

// CPU
void make_initial_fitness(int *host, int POPSIZE, int CHROMOSOME)
{
    for (int i = 0; i < POPSIZE; ++i)
    {
        host[i] = rand() % CHROMOSOME;
    }
}


int main(int argc, char **argv)
{
    // const int blocks = 1;
    // const int threads = 5;
    // const int N = blocks * threads;

    float elapsed_time = 0.0f;
    hipEvent_t start, end;

    int POPSIZE = 100;
    int CHROMOSOME = 128;
    // std::cout << argc << std::endl;
    if (argc == 3)
    {
        // std::cout << argv[1] << "," << argv[2] << std::endl;
        POPSIZE = std::atoi(argv[1]);
        CHROMOSOME = std::atoi(argv[2]);
    }

    std::cout << POPSIZE << "," << CHROMOSOME << std::endl;

    hipEventCreate(&start);
    hipEventCreate(&end);

    thrust::host_vector  <int> host_fit(POPSIZE);
    thrust::device_vector<int> dev_fit(POPSIZE);
    thrust::host_vector  <int> host_id(POPSIZE);
    thrust::device_vector<int> dev_id(POPSIZE);

    // イニシャライズ
    make_initial_fitness(thrust::raw_pointer_cast(&host_fit[0]), POPSIZE, CHROMOSOME);
    thrust::sequence(host_id.begin(), host_id.end());
    printf("### PRE ###\n");
    // show_host(host_id, host_fit);

    // コピー CPU --> GPU
    dev_fit = host_fit;
    dev_id  = host_id;

    // sort
    hipEventRecord(start, 0);
    thrust::sort_by_key(dev_fit.begin(), dev_fit.end(), dev_id.begin());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    printf("Elapsed Time(thrust sort) %f\n", elapsed_time);

    // コピー GPU --> CPU
    host_fit = dev_fit;
    host_id  = dev_id;
    printf("### POST ###\n");
    // show_host(host_id, host_fit);

    // thrust::counting_iterator<int> ci(1);
    // thrust::device_vector<int> d_x(ci, ci + N);
    // thrust::device_vector<int> d_o(1);

    /*
    int *pdev = thrust::raw_pointer_cast(&dev[0]);

    reduction1_kernel<int, N> <<<blocks, threads>>> (
            thrust::raw_pointer_cast(d_x.data()),
            thrust::raw_pointer_cast(d_o.data())
            );

    thrust::copy(thrust::host, d_o.cbegin(), d_o.cend(), std::ostream_iterator<int>(std::cout, " "));
    */
    return 0;
}


